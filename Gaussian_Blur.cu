
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdbool.h>
#include <stdio.h>
#include <cmath>
#include <iostream>
using namespace std;

#define BLOCKSIZE 1024
#define TILEWIDTH 32
#define RADIUS 1
#define SIGMA 1.
__constant__ float MASK[RADIUS+1];
__constant__ float DENOM;

typedef struct {
  union { int width, w; }; // width of image
  union { int height, h; }; // height of image
  union { uint8_t *pixels, *p; }; // pixel bgr values (rgb, reversed)
  union { uint8_t bytesPerPixel, bpp; }; // number of bytes in each pixel (3 or 4)
} sprite;

/** loads .bmp into provided sprite */
extern "C" int loadFile(sprite *sprite, const char *filename);

/** writes provided sprite into new bmp */
extern "C" bool writeFile(sprite *sprite, const char *writeFile);

/** provides the absolute value of an input */
__device__ int pos(int number) {
  int out = number & 0x7FFFFFFF; // fancy bitfuckery erases sign bit
  return out; // now sign bit is 0!!
}

/** 
Performs a blur operation on the input using the provided convolution kernel.
  Uses 1D symmetrical convolution, which gets applied horizontally ONLY in this step.
  The next step will calculate the identical convolution in the vertical direction.

@param input: The input image
@param output: Where to put the blurred image
@param w, @param h, @param depth: The sprite information for use in kernel
@param r: The 1D Gaussian kernel and its radius. 
  Kernel submitted through constant memory, uses ONLY first r+1 elements
@param vertical: Is the input coalesced or should we stride by h within each block?
*/
__global__ void gaussianBlurLine(uint8_t* input, uint8_t* output, // in- and out-puts
				   const int w, const int h, const int depth, // width, height, and depth of the image
				   const int rad, // kernel radius
				   const bool vertical // are we reading vertically or horizontally
				   ){
  // memory caches
  int radw = rad; if (vertical) { radw *= h; }
  __shared__ uint8_t red[BLOCKSIZE + 2*RADIUS]; // shared arrays with halo.
  __shared__ uint8_t green[BLOCKSIZE + 2*RADIUS]; // greens
  __shared__ uint8_t blue[BLOCKSIZE + 2*RADIUS]; // blues
  int lindex = threadIdx.x + rad;

  if (lindex-rad < w && blockIdx.x < h){
    // for now, block size has to be at least h
    int gindex = vertical ? depth * (blockIdx.x + threadIdx.x*w) : depth * (blockIdx.x*w + threadIdx.x); 
    
    // load into shared memory
    uint8_t b = input[gindex], g = input[gindex+1], r = input[gindex+2];
    blue[lindex] = b; green[lindex] = g; red[lindex] = r;

    // halo handling
    if (lindex < 2*rad) {
      int hindex = gindex - (radw*depth); // halo index
      if (hindex < 0) { // use what we've already read
	int iout = 2*rad - lindex - 1;
	blue[iout] = b; green[iout] = g; red[iout] = r; // no need to read new vals
      } else { // play fetch
	uint8_t bh = input[hindex], gh = input[hindex+1], rh = input[hindex+2]; // handle halos normally
	blue[lindex-rad] = bh; green[lindex-rad+1] = gh; red[lindex-rad+2] = rh;
      }
    }
    if (lindex >= blockDim.x) { // these are separated to handle r's greater than half the block size.
      int hindex = gindex + (radw*depth); // halo index
      if (hindex >= w) { // swap h and w on second call
	int mod = w-1 - lindex+rad; // in 0..(r-1), where highest possible = 0, lowest =(r-1)
	int iout = w+rad + mod;
	blue[iout] = b; green[iout] = g; red[iout] = r;
      } else {
	uint8_t bh = input[hindex], gh = input[hindex+1], rh = input[hindex+2]; // handle halos normally
	blue[lindex+rad] = bh; green[lindex+rad+1] = gh; red[lindex+rad+2] = rh;
      }
    }
    __syncthreads(); // patience

    // math
    float rSum = 0, gSum = 0, bSum = 0;
    for (int i = -rad; i <= rad; i++) {
      /** calculate stuff */
      float f = 1. / (2*r+1); // {-r, r}->0, 0->r, correct order in between.
      rSum += red[lindex+i] *f;
      gSum += green[lindex+i] *f;
      bSum += blue[lindex+i] *f;
    }

    // write
    output[gindex] = (uint8_t) bSum;
    output[gindex+1] = (uint8_t) gSum;
    output[gindex+2] = (uint8_t) rSum;
  }
}

/**
Generates the first half +1 elements of a Gaussian kernel
Because Gaussian kernels are symmetric, this can be extrapolated to a full kernel, and will be later.
Done this way because there's fewer CPU math operations this way T-T
  and also i'm a massochist or smth idk
 */
__host__ float* gaussianKernel(const int r, const float sigma) {
  float* out = (float*) malloc ( (r+1)*sizeof(float) );
  float s = 2*sigma*sigma;
  
  float sum = 0.; // for normalizing
  for (int x = -r; x <= 0; x++) { // only use first half of kernel for calculations
    out[x+r] = exp(-(x*x) / s) / (M_PI * s);
    sum += x==0 ? out[x+r] : 2*out[x+r];
  }

  for (int i = 0; i <= r; i++) {
    out[i] /= sum; // normalize
  }

  return out;
}

/** Performs a blur operation on the input using a flat 1/r^2 convolution kernel */
__host__ float* flatKernel(const int r) {
  float* out = (float*) malloc ( (r+1)*sizeof(float) );

  for (int i = 0; i <=r; i++) {
    out[i] = 1 / (2*r+1);
  }

  return out;
}

/**
Runs a full Gaussian blur, start to finish, including copying out from cuda into the provided sprite's pixel list.
 */
__host__ bool gaussianBlur(sprite* sprite, const int r, const float sig) {
  if (sprite->w > BLOCKSIZE || sprite->h > BLOCKSIZE) {
    cerr << "Unable to blur images greater than " << BLOCKSIZE << " in a single dimension.\n";
    return false;
  } if (r >= sprite->h || r >= sprite->w) {
    cerr << "Unable to blur images with r > either image dimension.\n";
    return false;
  }
  
  // cuda malloc
  int size = sprite->w * sprite->h * sprite->bpp;
  uint8_t *in_pixels, *out_pixels;
  hipMalloc(&in_pixels, sizeof(uint8_t) * size);
  hipMalloc(&out_pixels, sizeof(uint8_t) * size);
  hipMemcpy(in_pixels, sprite->p, size, hipMemcpyHostToDevice);

  // run kernel
  float* mask = flatKernel(r);
  hipMemcpyToSymbol(HIP_SYMBOL(MASK), mask, (r+1)*sizeof(float));
  //int blockX = ceil ( (1.*sprite.w) / TILEWIDTH ), blockY = ceil ( (1.*sprite.h) / TILEWIDTH );
  //int xWidth = TILEWIDTH < sprite.w ? TILEWIDTH : sprite.w, yWidth = TILEWIDTH < sprite.h ? TILEWIDTH : sprite.h;
  //dim3 dimGrid( blockX, blockY, 1);
  //dim3 dimBlock( xWidth, yWidth, 1);
  gaussianBlurLine<<<sprite->h, sprite->w>>>(in_pixels, out_pixels,
					     sprite->w, sprite->h, sprite->bpp,
					     r, false);
  hipDeviceSynchronize(); // IMMEDIATELY after opening the kernel >:|
  //gaussianBlurLine<<<sprite->w, sprite->h>>>(out_pixels, in_pixels, // swap so output goes back in
  //				     sprite->h, sprite->w, sprite->bpp,
  //					     r, true);
  //cudaDeviceSynchronize();
  cerr << hipGetErrorString(hipGetLastError()) << "\n";

  // write file out
  hipMemcpy(sprite->p, in_pixels, size, hipMemcpyDeviceToHost);

  // freedom!!
  free(mask);
  hipFree(in_pixels);
  hipFree(out_pixels);
  return true;
}

int main(int argc, char *argv[]) {
  // initialize sprite
  static sprite sprite;
  int pixels_read = loadFile(&sprite, argv[1]); cout << pixels_read << "\n";
  int size = pixels_read * sprite.bpp;

  bool success = gaussianBlur(&sprite, RADIUS, SIGMA);
  if (!success) {
    cerr << "Blur kernel failed. See above for more detail error messaging.\n";
    return -1;
  }
  
  bool wrote = writeFile(&sprite, "outputs/blur_test.bmp"); // TODO accept second CL arg

  for (int i = 0; i < size; i++) {
    cout << (int) sprite.p[i] << " "; // print output to make sure it looks right
  }
  cout << "\n";

  // freedom!!
  free(sprite.p);

  return 0;
}

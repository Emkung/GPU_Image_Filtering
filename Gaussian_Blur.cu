
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdbool.h>
#include <stdio.h>
#include <cmath>
#include <iostream>
using namespace std;

#define BLOCKSIZE 1024
#define TILEWIDTH 32
#define RADIUS 20
#define SIGMA 1.5
__constant__ float MASK[2*RADIUS+1];

typedef struct {
  union { int width, w; }; // width of image
  union { int height, h; }; // height of image
  union { uint8_t *pixels, *p; }; // pixel bgr values (rgb, reversed)
  union { uint8_t bytesPerPixel, bpp; }; // number of bytes in each pixel (3 or 4)
} sprite;

/** loads .bmp into provided sprite */
extern "C" int loadFile(sprite *sprite, const char *filename);

/** writes provided sprite into new bmp */
extern "C" bool writeFile(sprite *sprite, const char *writeFile);

/** provides the absolute value of an input */
__device__ int pos(int number) {
  int out = number & 0x7FFFFFFF; // fancy bitfuckery erases sign bit
  return out; // now sign bit is 0!!
}

/** 
Performs a blur operation on the input using the provided convolution kernel.
  Uses 1D symmetrical convolution, which gets applied horizontally ONLY in this step.
  The next step will calculate the identical convolution in the vertical direction.

@param input: The input image
@param output: Where to put the blurred image
@param w, @param h, @param depth: The sprite information for use in kernel
@param r: The 1D Gaussian kernel and its radius. 
  Kernel submitted through constant memory
@param vertical: Is the input coalesced or should we stride by h within each block?
*/
__global__ void gaussianBlurLine(uint8_t* input, uint8_t* output, // in- and out-puts
				   const int w, const int h, const int depth, // width, height, and depth of the image
				   const int rad, // kernel radius
				   const bool vertical // are we reading vertically or horizontally
				   ){
  // memory caches
  __shared__ uint8_t red[BLOCKSIZE + 2*RADIUS]; // shared arrays with halo.
  __shared__ uint8_t green[BLOCKSIZE + 2*RADIUS]; // greens
  __shared__ uint8_t blue[BLOCKSIZE + 2*RADIUS]; // blues
  int radw = rad; if (vertical) { radw *= h; }
  int lindex = threadIdx.x + rad;
  int line = blockIdx.x*blockDim.x % h; // which row or column is this block covering?
  int tile = blockIdx.x*blockDim.x / h; // how many blocks are above me?

  if (lindex-rad < w && blockIdx.x < h){
    // for now, block size has to be at least h
    int gindex = vertical ? depth * (line + h*(tile*h+threadIdx.x)) : depth * (line*w + (tile*h + threadIdx.x));
    
    
    // load into shared memory
    uint8_t b = input[gindex], g = input[gindex+1], r = input[gindex+2];
    blue[lindex] = b; green[lindex] = g; red[lindex] = r;

    // halo handling
    if (lindex < 2*rad) {
      int hindex = gindex - (radw*depth); // halo index
      if (hindex < 0) { // use what we've already read
	int iout = 2*rad - lindex - 1;
	blue[iout] = b; green[iout] = g; red[iout] = r; // no need to read new vals
      } else { // play fetch
	uint8_t bh = input[hindex], gh = input[hindex+1], rh = input[hindex+2]; // handle halos normally
	blue[lindex-rad] = bh; green[lindex-rad+1] = gh; red[lindex-rad+2] = rh;
      }
    }
    if (lindex >= blockDim.x) { // these are separated to handle r's greater than half the block size.
      int hindex = gindex + (radw*depth); // halo index
      if (hindex >= w) { // swap h and w on second call
	int mod = w-1 - lindex+rad; // in 0..(r-1), where highest possible = 0, lowest =(r-1)
	int iout = w+rad + mod;
	blue[iout] = b; green[iout] = g; red[iout] = r;
      } else {
	uint8_t bh = input[hindex], gh = input[hindex+1], rh = input[hindex+2]; // handle halos normally
	blue[lindex+rad] = bh; green[lindex+rad+1] = gh; red[lindex+rad+2] = rh;
      }
    }
    __syncthreads(); // patience

    // math
    float rSum = 0, gSum = 0, bSum = 0;
    for (int i = -rad; i <= rad; i++) {
      /** calculate stuff */
      float f = MASK[i+rad];
      rSum += static_cast<float>(red[lindex+i]) *f;
      gSum += static_cast<float>(green[lindex+i]) *f;
      bSum += static_cast<float>(blue[lindex+i]) *f;
    }

    // write
    output[gindex] = static_cast<uint8_t>(bSum);
    output[gindex+1] = static_cast<uint8_t>(gSum);
    output[gindex+2] = static_cast<uint8_t>(rSum);
  }
}

/** Generates a Gaussian kernel */
__host__ float* gaussianKernel(const int r, const float sigma) {
  float* out = (float*) malloc ( (2*r+1)*sizeof(float) );
  float s = 2*sigma*sigma;
  
  float sum = 0.; // for normalizing
  for (int x = -r; x <= r; x++) { // only use first half of kernel for calculations
    out[x+r] = exp(-(x*x) / s) / (M_PI * s);
    sum += out[x+r];
  }

  for (int i = 0; i <= 2*r; i++) {
    out[i] /= sum; // normalize
  }

  return out;
}

/** Generates a flat 1/(2r+1) convolution kernel */
__host__ float* flatKernel(const int r) {
  float* out = (float*) malloc ( (2*r+1)*sizeof(float) );

  for (int i = 0; i <= 2*r; i++) {
    out[i] = 1. / (2*r+1);
  }

  return out;
}

/**
Runs a full Gaussian blur, start to finish, including copying out from cuda into the provided sprite's pixel list.
 */
__host__ bool gaussianBlur(sprite* sprite, const int r, const float sig) {
  if (r >= sprite->h || r >= sprite->w) {
    cerr << "Unable to blur images with r > either image dimension.\n";
    return false;
  }
  
  // cuda malloc
  int size = sprite->w * sprite->h * sprite->bpp;
  uint8_t *in_pixels, *out_pixels;
  hipMalloc(&in_pixels, sizeof(uint8_t) * size);
  hipMalloc(&out_pixels, sizeof(uint8_t) * size);
  hipMemcpy(in_pixels, sprite->p, size, hipMemcpyHostToDevice);

  // run kernel
  float* mask = gaussianKernel(r, sig);
  hipMemcpyToSymbol(HIP_SYMBOL(MASK), mask, (2*r+1)*sizeof(float));
  int blocks = ceil ( (1.*sprite->w) / BLOCKSIZE ) *sprite->h;
  int threads = blocks > 1 ? BLOCKSIZE : sprite->w;
  gaussianBlurLine<<<blocks, threads>>>(in_pixels, out_pixels,
					sprite->w, sprite->h, sprite->bpp,
					r, false);
  blocks = ceil ( (1.*sprite->h) / BLOCKSIZE ) *sprite->w;
  threads = blocks > 1 ? BLOCKSIZE : sprite->h;
  hipDeviceSynchronize(); // trust me this saves time
  gaussianBlurLine<<<blocks, threads>>>(out_pixels, in_pixels, // swap so output goes back in
					sprite->h, sprite->w, sprite->bpp,
					r, true);
  //cudaDeviceSynchronize();
  cerr << hipGetErrorString(hipGetLastError()) << "\n";

  // write file out
  hipMemcpy(sprite->p, in_pixels, size, hipMemcpyDeviceToHost);

  // freedom!!
  free(mask);
  hipFree(in_pixels);
  hipFree(out_pixels);
  return true;
}

int main(int argc, char *argv[]) {
  // initialize sprite
  static sprite sprite;
  int pixels_read = loadFile(&sprite, argv[1]); cout << pixels_read << "\n";
  int size = pixels_read * sprite.bpp;

  bool success = gaussianBlur(&sprite, RADIUS, SIGMA);
  if (!success) {
    cerr << "Blur kernel failed. See above for more detail error messaging.\n";
    return -1;
  }
  
  bool wrote = writeFile(&sprite, "outputs/blur_big_test.bmp"); // TODO accept second CL arg

  for (int i = 0; i < size; i++) {
    cout << (int) sprite.p[i] << " "; // print output to make sure it looks right
  }
  cout << "\n";

  // freedom!!
  free(sprite.p);

  return 0;
}

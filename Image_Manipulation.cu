#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 250

int i;
int N;

 __global__ void MatrixMulOnDevice(float* A, float* B, float* C, int Width) {
   for (int i = 0; i < Width; ++i) {
     for (int j = 0; j < Width; ++j) {
       float sum = 0;
       for (int k = 0; k < Width; ++k) {
         float a = A[i * Width + k];
	 float b = B[k * Width + j];
	 sum += a * b;
       }
       C[i * Width + j] = sum;
     }
   }
}

int main() {

  int size = SIZE;

  float *x, *y, *z;
    hipMallocManaged(&x, SIZE*sizeof(float) * size * size);
    hipMallocManaged(&y, SIZE*sizeof(float) * size * size);
    hipMallocManaged(&z, SIZE*sizeof(float) * size * size);

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      x[i * size + j] = 1; // x[i][j]
      y[i * size + j] = 1;
    }
  printf("\n");
  }

  MatrixMulOnDevice<<<1, 128>>>(x, y, z, size);

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipDeviceSynchronize();

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      printf("%f ", z[i * size + j]);
      if (z[i * size + j] != size) {
        printf("Error at z[%d][%d]: %f\n", i, j, z[i * size + j]);
      }
    }
    printf("\n");
  }

  hipFree(x);
  hipFree(y);
  hipFree(z);

  double t1 = get_clock();
  printf("time per call: %f\n", t1 - t0);

  return 0;
}
